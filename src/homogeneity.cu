#include "hip/hip_runtime.h"

/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <tiffio.h>
#include <math.h>
#include <iostream>
#include <string.h>
#include <string>
#include <fstream>
#include <sstream>
#include <map>
#include <vector>
#include <algorithm>
#include <time.h>
#include <chrono>
#include <queue>
#include <set>
#include <hip/hip_runtime.h>

static const int WORK_SIZE = 256;
const int AGP = 14, PAS = 15, AGR = 18, CAP = 19, CSP = 20, MAP = 21;

#define CHECK_CUDA_RESULT(N) {											\
	hipError_t result = N;												\
	if (result != 0) {													\
		printf("CUDA call on line %d returned error %d\n", __LINE__,	\
			result);													\
		exit(1);														\
	} }

void setup(TIFF* new_tif, int width, int length, int bitsPerSample, int sampleFormat){

    TIFFSetField(new_tif, TIFFTAG_IMAGEWIDTH     , width);
    TIFFSetField(new_tif, TIFFTAG_IMAGELENGTH    , length);
    TIFFSetField(new_tif, TIFFTAG_BITSPERSAMPLE  , bitsPerSample);
    TIFFSetField(new_tif, TIFFTAG_SAMPLEFORMAT   , sampleFormat);
    TIFFSetField(new_tif, TIFFTAG_COMPRESSION    , 1);
    TIFFSetField(new_tif, TIFFTAG_PHOTOMETRIC    , 1);
    TIFFSetField(new_tif, TIFFTAG_SAMPLESPERPIXEL, 1);
    TIFFSetField(new_tif, TIFFTAG_ROWSPERSTRIP   , 1);
    TIFFSetField(new_tif, TIFFTAG_RESOLUTIONUNIT , 1);
    TIFFSetField(new_tif, TIFFTAG_XRESOLUTION    , 1);
    TIFFSetField(new_tif, TIFFTAG_YRESOLUTION    , 1);
    TIFFSetField(new_tif, TIFFTAG_PLANARCONFIG   , PLANARCONFIG_CONTIG);

};

void write_line_tiff(TIFF* tif, double tif_line[], int line){

    if (TIFFWriteScanline(tif, tif_line, line) < 0){
        std::cerr << "Write problem!" << std::endl;
        exit(4);
    }

};

void write_line_tiff(TIFF* tif, int tif_line[], int line){

    if (TIFFWriteScanline(tif, tif_line, line) < 0){
        std::cerr << "Write problem!" << std::endl;
        exit(4);
    }

};

void read_line_tiff(TIFF* tif, double tif_line[], int line){
    if(TIFFReadScanline(tif, tif_line, line) < 0){
        std::cerr << "Read problem" << std::endl;
        exit(3);
    }
};

void read_line_tiff(TIFF* tif, int tif_line[], int line){
    if(TIFFReadScanline(tif, tif_line, line) < 0){
        std::cerr << "Read problem" << std::endl;
        exit(3);
    }
};

__host__ __device__ bool checkLandCode(int value){

    return (value == AGP) || (value == PAS) || (value == AGR) || (value == CAP) || (value == CSP) || (value == MAP);

}

__global__ void landCoverHomogeneity(double* inputBuffer, int* output, int line, int numCol, int numLine){

	int column = threadIdx.x + blockIdx.x * blockDim.x;
	double pixel_value;
	int aux;

	while (column < numCol) {

		aux = line % 7;

		pixel_value = inputBuffer[aux * numCol + column];

		output[column] = false;

		if(checkLandCode(pixel_value)) { //Verify if the pixel is an AGR pixel

			output[column] = true;

			for(int i = -3; i <= 3 && output[column]; i++){

				for(int j = -3; j <= 3 && output[column]; j++){

					// Check if the neighbor is AGR too

					if (column + i >= 0 && column + i < numCol && line + j >= 0 && line + j < numLine) {

						aux = (line + j) % 7;

						pixel_value = inputBuffer[aux * numCol + column];

						if(!isnan(pixel_value))
							if(!checkLandCode(pixel_value))
								output[column] = false;

					}

				}

			}

		}

		column += blockDim.x * gridDim.x;

	}

}

void testLandCoverHomogeneity(TIFF* landCover, TIFF* mask){

    uint32 height_band, width_band;
    TIFFGetField(landCover, TIFFTAG_IMAGELENGTH, &height_band);
    TIFFGetField(landCover, TIFFTAG_IMAGEWIDTH, &width_band);

    double* buffer = (double *) malloc(7 * width_band * sizeof(double));

    int relation[7] = {-1, -1, -1, -1, -1, -1, -1}, aux;

    for(int line = 0; line < height_band; line++) {

        // Create the respective line of the binary map of eligibles pixels
        int mask_line[width_band];

        for(int column = 0; column < width_band; column++) {

            int pixel_value;

            aux = line % 7;

            if(relation[aux] != line) {

                read_line_tiff(landCover, buffer + aux * width_band, line);
                relation[aux] = line;

            }

            pixel_value = buffer[aux * width_band + column];

            mask_line[column] = false;

            if(checkLandCode(pixel_value)) { //Verify if the pixel is an AGR pixel

                mask_line[column] = true;

                for(int i = -3; i <= 3 && mask_line[column]; i++){

                    for(int j = -3; j <= 3 && mask_line[column]; j++){

                        // Check if the neighbor is AGR too

                        if (column + i >= 0 && column + i < width_band && line + j >= 0 && line + j < height_band) {

                            aux = (line + j) % 7;

                            if(relation[aux] != (line + j)) {

                                read_line_tiff(landCover, buffer + aux * width_band, line + j);
                                relation[aux] = (line + j);

                            }

                            pixel_value = buffer[aux * width_band + column];

                            if(!std::isnan(pixel_value))
                                if(!checkLandCode(pixel_value))
                                    mask_line[column] = false;

                        }

                    }

                }

            }

        }

        write_line_tiff(mask, mask_line, line);

    }

//    for(int i = 0; i < 7; i++){
//        free(buffer[i]);
//    }
    free(buffer);

}

int main(int argc, char **argv) {

	std::string landCoverPath = argv[1];
	std::string outputPath = argv[2];

	std::string outputCPU = outputPath + "/CPU.tif";
	std::string outputGPU = outputPath + "/GPU.tif";

	TIFF* landCover = TIFFOpen(landCoverPath.c_str(), "rm");

	uint32 height_band, width_band;

	TIFFGetField(landCover, TIFFTAG_IMAGEWIDTH, &width_band);
	TIFFGetField(landCover, TIFFTAG_IMAGELENGTH, &height_band);

	TIFF* CPU = TIFFOpen(outputCPU.c_str(), "w8m");
	setup(CPU, width_band, height_band, 32, 2);

	testLandCoverHomogeneity(landCover, CPU);

	TIFFClose(CPU);

	TIFF* GPU = TIFFOpen(outputGPU.c_str(), "w8m");
	setup(GPU, width_band, height_band, 32, 2);

	double* buffer = (double *) malloc(7 * width_band * sizeof(double));
	int* output_line = (int*) malloc(width_band * sizeof(int));

	double* buffer_dev;
	hipMalloc((void**) &buffer_dev, 7 * width_band * sizeof(double*));

	int* output_dev;
	hipMalloc((void**) &output_dev, width_band * sizeof(int*));

	int relation[7] = {-1, -1, -1, -1, -1, -1, -1};

	for(int line = 0; line < height_band; line++) {

		for(int i = -3; i < 4; i++) {

			if(line + i >= 0 && line + i < height_band){

				if(relation[(line + i) % 7] != (line + i)) {
					read_line_tiff(landCover, buffer + ((line + i) % 7) * width_band, line + i);
					relation[(line + i) % 7] = line + i;
				}

			}

		}

		hipMemcpy(buffer_dev, buffer, 7 * width_band * sizeof(double), hipMemcpyHostToDevice);

		landCoverHomogeneity<<< (width_band + 1) / WORK_SIZE , WORK_SIZE>>>(buffer_dev, output_dev, line, width_band, height_band);

		hipMemcpy(output_line, output_dev, width_band * sizeof(int), hipMemcpyDeviceToHost);

		write_line_tiff(GPU, output_line, line);

	}

	free(buffer);
	free(output_line);
	hipFree(buffer_dev);
	hipFree(output_dev);

	TIFFClose(landCover);
	TIFFClose(GPU);

	return 0;
}
